#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello(){
    printf("Hello CUDA World !!\n");
}

int main() {
    hello<<< 2, 4 >>>();
    hipDeviceSynchronize();
    return 0;
}