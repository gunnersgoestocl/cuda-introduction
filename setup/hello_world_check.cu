#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDAエラーチェック用マクロ
#define CUDA_CHECK(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA Error: %s at line %d in file %s\n", \
               hipGetErrorString(err), __LINE__, __FILE__); \
        exit(1); \
    } \
} while(0)

__global__ void hello(){
    printf("Hello CUDA World !!\n");
}

int main() {
    hello<<< 2, 4 >>>();
    // カーネル起動エラーをチェック
    CUDA_CHECK(hipGetLastError());
    // 同期エラーもチェック
    CUDA_CHECK(hipDeviceSynchronize());
    return 0;
}