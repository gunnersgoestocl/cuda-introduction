#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    
    if (error != hipSuccess) {
        printf("Error getting device count: %s\n", hipGetErrorString(error));
        return 1;
    }
    
    printf("CUDA Device Count: %d\n", deviceCount);
    
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);
        
        printf("Device %d: %s\n", i, deviceProp.name);
        printf("  Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
        printf("  Total global memory: %.2f GB\n", deviceProp.totalGlobalMem / (1024.0 * 1024.0 * 1024.0));
    }
    
    return 0;
}