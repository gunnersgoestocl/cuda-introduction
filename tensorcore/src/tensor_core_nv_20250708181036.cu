#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <chrono>

// CUDA関連のヘッダー
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

// TensorCoreのブロックサイズ
#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

// 行列サイズ - TensorCoreに最適化された16の倍数
#define M 1024
#define N 1024
#define K 1024

// CUDA計測用マクロ
#define CUDA_CHECK(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

// 詳細な計測結果を格納する構造体
typedef struct {
    float total_time;            // 全体の実行時間
    float copy_in_time;          // ホスト→デバイスのコピー時間
    float kernel_time;           // カーネル実行時間
    float global_to_frag_time;   // グローバルメモリ→フラグメント時間
    float global_to_shared_time; // グローバルメモリ→共有メモリ時間
    float shared_to_frag_time;   // 共有メモリ→フラグメント時間
    float compute_time;          // 計算時間
    float store_time;            // 結果の書き戻し時間
    float copy_out_time;         // デバイス→ホストのコピー時間
    float sync_overhead;         // 同期オーバーヘッド
} TimingInfo;

// 時間計測結果を格納する構造体（デバイス用）
typedef struct {
    uint64_t global_to_frag_start;
    uint64_t global_to_frag_end;
    uint64_t global_to_shared_start;
    uint64_t global_to_shared_end;
    uint64_t shared_to_frag_start;
    uint64_t shared_to_frag_end;
    uint64_t compute_start;
    uint64_t compute_end;
    uint64_t store_start;
    uint64_t store_end;
    uint64_t sync1_start;
    uint64_t sync1_end;
    uint64_t sync2_start;
    uint64_t sync2_end;
} DeviceTimingInfo;

// 行列初期化
void init_matrices(half *A, half *B, float *C) {
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < K; j++) {
            A[i * K + j] = __float2half((rand() % 10) / 10.0f);
        }
    }

    for (int i = 0; i < K; i++) {
        for (int j = 0; j < N; j++) {
            B[i * N + j] = __float2half((rand() % 10) / 10.0f);
        }
    }

    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            C[i * N + j] = 0.0f;
        }
    }
}

// 結果検証
bool validate_results(float *C, float *C_ref) {
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            if (fabs(C[i * N + j] - C_ref[i * N + j]) > 1e-1) {
                printf("Mismatch at (%d, %d): GPU=%f, CPU=%f\n", 
                       i, j, C[i * N + j], C_ref[i * N + j]);
                return false;
            }
        }
    }
    return true;
}

// グローバルメモリからフラグメントへ直接ロードする方式のカーネル（clock64で計測）
__global__ void tensor_core_global_memory_kernel(half *A, half *B, float *C, 
                                               int m, int n, int k,
                                               DeviceTimingInfo *timing_info) {
    // ブロックの位置を計算
    const int block_row = blockIdx.y;
    const int block_col = blockIdx.x;
    
    // グローバルメモリ上での開始位置を計算
    const int row_start = block_row * WMMA_M;
    const int col_start = block_col * WMMA_N;
    
    // このブロックが担当する行列の範囲がサイズ内にあるか確認
    if (row_start >= m || col_start >= n) return;
    
    // 時間計測用フラグ（スレッド0のみが計測）
    bool is_measuring_thread = (threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0);
    
    // フラグメント宣言
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, nvcuda::wmma::row_major> a_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, nvcuda::wmma::col_major> b_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
    
    // 結果フラグメントを0で初期化
    nvcuda::wmma::fill_fragment(c_frag, 0.0f);
    
    // K次元の各ステップでデータを計算
    for (int i = 0; i < k; i += WMMA_K) {
        if (i + WMMA_K > k) break;
        
        // グローバルメモリからフラグメントへのロード時間測定開始
        if (is_measuring_thread && i == 0) {
            timing_info->global_to_frag_start = clock64();
        }
        
        // Aフラグメントをグローバルメモリから直接ロード
        nvcuda::wmma::load_matrix_sync(a_frag, A + (row_start * k + i), k);
        
        // Bフラグメントをグローバルメモリから直接ロード
        nvcuda::wmma::load_matrix_sync(b_frag, B + (i * n + col_start), n);
        
        // グローバルメモリからフラグメントへのロード時間測定終了
        if (is_measuring_thread && i == 0) {
            timing_info->global_to_frag_end = clock64();
        }
        
        // 計算時間測定開始
        if (is_measuring_thread && i == 0) {
            timing_info->compute_start = clock64();
        }
        
        // 行列乗算を実行（TensorCore命令を使用）
        nvcuda::wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
        
        // 計算時間測定終了
        if (is_measuring_thread && i == 0) {
            timing_info->compute_end = clock64();
        }
    }
    
    // 結果の書き戻し時間測定開始
    if (is_measuring_thread) {
        timing_info->store_start = clock64();
    }
    
    // 結果をグローバルメモリに書き戻す
    nvcuda::wmma::store_matrix_sync(C + (row_start * n + col_start), c_frag, n, nvcuda::wmma::mem_row_major);
    
    // 結果の書き戻し時間測定終了
    if (is_measuring_thread) {
        timing_info->store_end = clock64();
    }
}

// 共有メモリを使用したTensorCoreカーネル（正しいパディング実装）
__global__ void tensor_core_shared_memory_kernel(half *A, half *B, float *C, 
                                               int m, int n, int k,
                                               DeviceTimingInfo *timing_info) {
    // 共有メモリのパディング（16バイトアライメントを維持）
    // パディングサイズをWMMA_Kの倍数に調整
    const int padded_k = (WMMA_K + 8) & ~0xF; // 16の倍数になるようにパディング
    const int padded_n = (WMMA_N + 8) & ~0xF; // 16の倍数になるようにパディング
    
    // アライメントを保ったパディング付き共有メモリ
    __shared__ half shared_A[WMMA_M][padded_k];
    __shared__ half shared_B[WMMA_K][padded_n];
    
    // ブロックの位置を計算
    const int block_row = blockIdx.y;
    const int block_col = blockIdx.x;
    
    // グローバルメモリ上での開始位置を計算
    const int row_start = block_row * WMMA_M;
    const int col_start = block_col * WMMA_N;
    
    // このブロックが担当する行列の範囲がサイズ内にあるか確認
    if (row_start >= m || col_start >= n) return;
    
    // 時間計測用フラグ（スレッド0のみが計測）
    bool is_measuring_thread = (threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0);
    
    // スレッドID
    const int tid = threadIdx.x;
    
    // フラグメント宣言
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, nvcuda::wmma::row_major> a_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, nvcuda::wmma::col_major> b_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
    
    // 結果フラグメントを0で初期化
    nvcuda::wmma::fill_fragment(c_frag, 0.0f);
    
    // K次元の各ステップでデータを計算
    for (int i = 0; i < k; i += WMMA_K) {
        if (i + WMMA_K > k) break;
        
        // 共有メモリの初期化（オプション - デバッグに役立つ）
        for (int j = tid; j < WMMA_M * padded_k; j += blockDim.x) {
            int row = j / padded_k;
            int col = j % padded_k;
            shared_A[row][col] = __float2half(0.0f);
        }
        
        for (int j = tid; j < WMMA_K * padded_n; j += blockDim.x) {
            int row = j / padded_n;
            int col = j % padded_n;
            shared_B[row][col] = __float2half(0.0f);
        }

        // グローバルメモリから共有メモリへのロード時間測定開始
        if (is_measuring_thread && i == 0) {
            timing_info->global_to_shared_start = clock64();
        }
        
        // 各スレッドが複数の要素をロード
        for (int j = tid; j < WMMA_M * WMMA_K; j += blockDim.x) {
            int row = j / WMMA_K;
            int col = j % WMMA_K;
            
            // A行列をグローバルメモリから共有メモリにロード
            if (row_start + row < m && i + col < k) {
                shared_A[row][col] = A[(row_start + row) * k + (i + col)];
            }
        }
        
        for (int j = tid; j < WMMA_K * WMMA_N; j += blockDim.x) {
            int row = j / WMMA_N;
            int col = j % WMMA_N;
            
            // B行列をグローバルメモリから共有メモリにロード
            if (i + row < k && col_start + col < n) {
                shared_B[row][col] = B[(i + row) * n + (col_start + col)];
            }
        }
        
        // グローバルメモリから共有メモリへのロード時間測定終了
        if (is_measuring_thread && i == 0) {
            timing_info->global_to_shared_end = clock64();
        }
        
        // 同期1時間測定開始
        if (is_measuring_thread && i == 0) {
            timing_info->sync1_start = clock64();
        }
        
        // 同期 - すべてのスレッドがデータをロードするまで待機
        __syncthreads();
        
        // 同期1時間測定終了
        if (is_measuring_thread && i == 0) {
            timing_info->sync1_end = clock64();
        }
        
        // 共有メモリからフラグメントへのロード時間測定開始
        if (is_measuring_thread && i == 0) {
            timing_info->shared_to_frag_start = clock64();
        }
        
        // 共有メモリからフラグメントにロード - ストライドに正しいパディング値を使用
        nvcuda::wmma::load_matrix_sync(a_frag, &shared_A[0][0], padded_k);
        nvcuda::wmma::load_matrix_sync(b_frag, &shared_B[0][0], padded_n);
        
        // 共有メモリからフラグメントへのロード時間測定終了
        if (is_measuring_thread && i == 0) {
            timing_info->shared_to_frag_end = clock64();
        }
        
        // 計算時間測定開始
        if (is_measuring_thread && i == 0) {
            timing_info->compute_start = clock64();
        }
        
        // 行列乗算を実行（TensorCore命令を使用）
        nvcuda::wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
        
        // 計算時間測定終了
        if (is_measuring_thread && i == 0) {
            timing_info->compute_end = clock64();
        }
        
        // 同期2時間測定開始
        if (is_measuring_thread && i == 0) {
            timing_info->sync2_start = clock64();
        }
        
        // 同期 - 次のタイルをロードする前にすべての計算が完了するまで待機
        __syncthreads();
        
        // 同期2時間測定終了
        if (is_measuring_thread && i == 0) {
            timing_info->sync2_end = clock64();
        }
    }
    
    // 結果の書き戻し時間測定開始
    if (is_measuring_thread) {
        timing_info->store_start = clock64();
    }
    
    // 結果をグローバルメモリに書き戻す
    nvcuda::wmma::store_matrix_sync(C + (row_start * n + col_start), c_frag, n, nvcuda::wmma::mem_row_major);
    
    // 結果の書き戻し時間測定終了
    if (is_measuring_thread) {
        timing_info->store_end = clock64();
    }
}

// グローバルメモリ版プロファイリング関数
TimingInfo profile_tensor_core_global(half *h_A, half *h_B, float *h_C) {
    half *d_A, *d_B;
    float *d_C;
    TimingInfo timing = {0};
    
    // イベント作成
    hipEvent_t start_total, stop_total;
    hipEvent_t start_copy_in, stop_copy_in;
    hipEvent_t start_kernel, stop_kernel;
    hipEvent_t start_copy_out, stop_copy_out;
    
    CUDA_CHECK(hipEventCreate(&start_total));
    CUDA_CHECK(hipEventCreate(&stop_total));
    CUDA_CHECK(hipEventCreate(&start_copy_in));
    CUDA_CHECK(hipEventCreate(&stop_copy_in));
    CUDA_CHECK(hipEventCreate(&start_kernel));
    CUDA_CHECK(hipEventCreate(&stop_kernel));
    CUDA_CHECK(hipEventCreate(&start_copy_out));
    CUDA_CHECK(hipEventCreate(&stop_copy_out));
    
    // 全体の時間計測開始
    CUDA_CHECK(hipEventRecord(start_total));
    
    // GPUメモリ確保
    CUDA_CHECK(hipMalloc(&d_A, M * K * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_B, K * N * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_C, M * N * sizeof(float)));
    
    // 時間計測用構造体のメモリ確保
    DeviceTimingInfo *d_timing;
    DeviceTimingInfo h_timing;
    CUDA_CHECK(hipMalloc(&d_timing, sizeof(DeviceTimingInfo)));
    
    // データ転送時間計測開始
    CUDA_CHECK(hipEventRecord(start_copy_in));
    
    // データをGPUにコピー
    CUDA_CHECK(hipMemcpy(d_A, h_A, M * K * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, K * N * sizeof(half), hipMemcpyHostToDevice));
    
    // データ転送時間計測終了
    CUDA_CHECK(hipEventRecord(stop_copy_in));
    CUDA_CHECK(hipEventSynchronize(stop_copy_in));
    
    // カーネル設定
    dim3 grid((N + WMMA_N - 1) / WMMA_N, (M + WMMA_M - 1) / WMMA_M);
    dim3 block(32);  // TensorCoreはワープ単位で動作するため32スレッド
    
    // カーネル実行時間計測開始
    CUDA_CHECK(hipEventRecord(start_kernel));
    
    // カーネル実行
    tensor_core_global_memory_kernel<<<grid, block>>>(d_A, d_B, d_C, M, N, K, d_timing);
    
    // カーネル実行時間計測終了
    CUDA_CHECK(hipEventRecord(stop_kernel));
    CUDA_CHECK(hipEventSynchronize(stop_kernel));
    
    // 時間計測データをホストに転送
    CUDA_CHECK(hipMemcpy(&h_timing, d_timing, sizeof(DeviceTimingInfo), hipMemcpyDeviceToHost));
    
    // 結果転送時間計測開始
    CUDA_CHECK(hipEventRecord(start_copy_out));
    
    // 結果をホストにコピー
    CUDA_CHECK(hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));
    
    // 結果転送時間計測終了
    CUDA_CHECK(hipEventRecord(stop_copy_out));
    CUDA_CHECK(hipEventSynchronize(stop_copy_out));
    
    // 全体の時間計測終了
    CUDA_CHECK(hipEventRecord(stop_total));
    CUDA_CHECK(hipEventSynchronize(stop_total));
    
    // 時間を計算
    CUDA_CHECK(hipEventElapsedTime(&timing.total_time, start_total, stop_total));
    CUDA_CHECK(hipEventElapsedTime(&timing.copy_in_time, start_copy_in, stop_copy_in));
    CUDA_CHECK(hipEventElapsedTime(&timing.kernel_time, start_kernel, stop_kernel));
    CUDA_CHECK(hipEventElapsedTime(&timing.copy_out_time, start_copy_out, stop_copy_out));
    
    // GPUのクロックレートを取得（近似値としてプロパティから取得）
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    float clock_rate = prop.clockRate * 1000.0f; // Hz単位
    
    // clock64()の時間をミリ秒に変換
    timing.global_to_frag_time = 
        (float)(h_timing.global_to_frag_end - h_timing.global_to_frag_start) / clock_rate * 1000.0f;
    timing.compute_time = 
        (float)(h_timing.compute_end - h_timing.compute_start) / clock_rate * 1000.0f;
    timing.store_time = 
        (float)(h_timing.store_end - h_timing.store_start) / clock_rate * 1000.0f;
    
    // リソース解放
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    CUDA_CHECK(hipFree(d_timing));
    
    CUDA_CHECK(hipEventDestroy(start_total));
    CUDA_CHECK(hipEventDestroy(stop_total));
    CUDA_CHECK(hipEventDestroy(start_copy_in));
    CUDA_CHECK(hipEventDestroy(stop_copy_in));
    CUDA_CHECK(hipEventDestroy(start_kernel));
    CUDA_CHECK(hipEventDestroy(stop_kernel));
    CUDA_CHECK(hipEventDestroy(start_copy_out));
    CUDA_CHECK(hipEventDestroy(stop_copy_out));
    
    // ミリ秒から秒に変換
    timing.total_time /= 1000.0f;
    timing.copy_in_time /= 1000.0f;
    timing.kernel_time /= 1000.0f;
    timing.copy_out_time /= 1000.0f;
    timing.global_to_frag_time /= 1000.0f;
    timing.compute_time /= 1000.0f;
    timing.store_time /= 1000.0f;
    
    return timing;
}

// 共有メモリ版プロファイリング関数
TimingInfo profile_tensor_core_shared(half *h_A, half *h_B, float *h_C) {
    half *d_A, *d_B;
    float *d_C;
    TimingInfo timing = {0};
    
    // イベント作成
    hipEvent_t start_total, stop_total;
    hipEvent_t start_copy_in, stop_copy_in;
    hipEvent_t start_kernel, stop_kernel;
    hipEvent_t start_copy_out, stop_copy_out;
    
    CUDA_CHECK(hipEventCreate(&start_total));
    CUDA_CHECK(hipEventCreate(&stop_total));
    CUDA_CHECK(hipEventCreate(&start_copy_in));
    CUDA_CHECK(hipEventCreate(&stop_copy_in));
    CUDA_CHECK(hipEventCreate(&start_kernel));
    CUDA_CHECK(hipEventCreate(&stop_kernel));
    CUDA_CHECK(hipEventCreate(&start_copy_out));
    CUDA_CHECK(hipEventCreate(&stop_copy_out));
    
    // 全体の時間計測開始
    CUDA_CHECK(hipEventRecord(start_total));
    
    // GPUメモリ確保
    CUDA_CHECK(hipMalloc(&d_A, M * K * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_B, K * N * sizeof(half)));
    CUDA_CHECK(hipMalloc(&d_C, M * N * sizeof(float)));
    
    // 時間計測用構造体のメモリ確保
    DeviceTimingInfo *d_timing;
    DeviceTimingInfo h_timing;
    CUDA_CHECK(hipMalloc(&d_timing, sizeof(DeviceTimingInfo)));
    
    // データ転送時間計測開始
    CUDA_CHECK(hipEventRecord(start_copy_in));
    
    // データをGPUにコピー
    CUDA_CHECK(hipMemcpy(d_A, h_A, M * K * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, K * N * sizeof(half), hipMemcpyHostToDevice));
    
    // データ転送時間計測終了
    CUDA_CHECK(hipEventRecord(stop_copy_in));
    CUDA_CHECK(hipEventSynchronize(stop_copy_in));
    
    // カーネル設定
    dim3 grid((N + WMMA_N - 1) / WMMA_N, (M + WMMA_M - 1) / WMMA_M);
    dim3 block(32);  // TensorCoreはワープ単位で動作するため32スレッド
    
    // カーネル実行時間計測開始
    CUDA_CHECK(hipEventRecord(start_kernel));
    
    // カーネル実行
    tensor_core_shared_memory_kernel<<<grid, block>>>(d_A, d_B, d_C, M, N, K, d_timing);
    
    // カーネル実行時間計測終了
    CUDA_CHECK(hipEventRecord(stop_kernel));
    CUDA_CHECK(hipEventSynchronize(stop_kernel));
    
    // 時間計測データをホストに転送
    CUDA_CHECK(hipMemcpy(&h_timing, d_timing, sizeof(DeviceTimingInfo), hipMemcpyDeviceToHost));
    
    // 結果転送時間計測開始
    CUDA_CHECK(hipEventRecord(start_copy_out));
    
    // 結果をホストにコピー
    CUDA_CHECK(hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));
    
    // 結果転送時間計測終了
    CUDA_CHECK(hipEventRecord(stop_copy_out));
    CUDA_CHECK(hipEventSynchronize(stop_copy_out));
    
    // 全体の時間計測終了
    CUDA_CHECK(hipEventRecord(stop_total));
    CUDA_CHECK(hipEventSynchronize(stop_total));
    
    // 時間を計算
    CUDA_CHECK(hipEventElapsedTime(&timing.total_time, start_total, stop_total));
    CUDA_CHECK(hipEventElapsedTime(&timing.copy_in_time, start_copy_in, stop_copy_in));
    CUDA_CHECK(hipEventElapsedTime(&timing.kernel_time, start_kernel, stop_kernel));
    CUDA_CHECK(hipEventElapsedTime(&timing.copy_out_time, start_copy_out, stop_copy_out));
    
    // GPUのクロックレートを取得（近似値としてプロパティから取得）
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    float clock_rate = prop.clockRate * 1000.0f; // Hz単位
    
    // clock64()の時間をミリ秒に変換
    timing.global_to_shared_time = 
        (float)(h_timing.global_to_shared_end - h_timing.global_to_shared_start) / clock_rate * 1000.0f;
    timing.shared_to_frag_time = 
        (float)(h_timing.shared_to_frag_end - h_timing.shared_to_frag_start) / clock_rate * 1000.0f;
    timing.compute_time = 
        (float)(h_timing.compute_end - h_timing.compute_start) / clock_rate * 1000.0f;
    timing.store_time = 
        (float)(h_timing.store_end - h_timing.store_start) / clock_rate * 1000.0f;
    timing.sync_overhead = 
        (float)((h_timing.sync1_end - h_timing.sync1_start) + 
                (h_timing.sync2_end - h_timing.sync2_start)) / clock_rate * 1000.0f;
    
    // リソース解放
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    CUDA_CHECK(hipFree(d_timing));
    
    CUDA_CHECK(hipEventDestroy(start_total));
    CUDA_CHECK(hipEventDestroy(stop_total));
    CUDA_CHECK(hipEventDestroy(start_copy_in));
    CUDA_CHECK(hipEventDestroy(stop_copy_in));
    CUDA_CHECK(hipEventDestroy(start_kernel));
    CUDA_CHECK(hipEventDestroy(stop_kernel));
    CUDA_CHECK(hipEventDestroy(start_copy_out));
    CUDA_CHECK(hipEventDestroy(stop_copy_out));
    
    // ミリ秒から秒に変換
    timing.total_time /= 1000.0f;
    timing.copy_in_time /= 1000.0f;
    timing.kernel_time /= 1000.0f;
    timing.copy_out_time /= 1000.0f;
    timing.global_to_shared_time /= 1000.0f;
    timing.shared_to_frag_time /= 1000.0f;
    timing.compute_time /= 1000.0f;
    timing.store_time /= 1000.0f;
    timing.sync_overhead /= 1000.0f;
    
    return timing;
}

// CPU上での行列乗算（リファレンス実装）
double cpu_matmul(half *A, half *B, float *C) {
    auto start = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            float sum = 0.0f;
            for (int k = 0; k < K; k++) {
                sum += __half2float(A[i * K + k]) * __half2float(B[k * N + j]);
            }
            C[i * N + j] = sum;
        }
    }

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    return elapsed.count();
}

// タイムラインデータ出力関数を修正
void output_timeline_data(const TimingInfo& global_timing, const TimingInfo& shared_timing) {
    std::cout << "\n=== TIMELINE_DATA_START ===\n";
    
    // グローバルメモリ版のタイムライン（開始時刻を0とする）
    float global_start = 0.0f;
    float global_copy_in_end = global_timing.copy_in_time * 1000;
    float global_kernel_start = global_copy_in_end;
    float global_global_to_frag_end = global_kernel_start + global_timing.global_to_frag_time * 1000;
    float global_compute_end = global_global_to_frag_end + global_timing.compute_time * 1000;
    float global_store_end = global_compute_end + global_timing.store_time * 1000;
    float global_kernel_end = global_kernel_start + global_timing.kernel_time * 1000;
    float global_copy_out_end = global_kernel_end + global_timing.copy_out_time * 1000;
    
    std::cout << "PHASE,Global,H2D_Copy," << global_start << "," << global_copy_in_end << "\n";
    std::cout << "PHASE,Global,Global_to_Frag," << global_kernel_start << "," << global_global_to_frag_end << "\n";
    std::cout << "PHASE,Global,Compute," << global_global_to_frag_end << "," << global_compute_end << "\n";
    std::cout << "PHASE,Global,Store," << global_compute_end << "," << global_store_end << "\n";
    std::cout << "PHASE,Global,D2H_Copy," << global_kernel_end << "," << global_copy_out_end << "\n";
    
    // 共有メモリ版のタイムライン（開始時刻を0とする）
    float shared_start = 0.0f;
    float shared_copy_in_end = shared_timing.copy_in_time * 1000;
    float shared_kernel_start = shared_copy_in_end;
    float shared_global_to_shared_end = shared_kernel_start + shared_timing.global_to_shared_time * 1000;
    float shared_shared_to_frag_end = shared_global_to_shared_end + shared_timing.shared_to_frag_time * 1000;
    float shared_compute_end = shared_shared_to_frag_end + shared_timing.compute_time * 1000;
    float shared_store_end = shared_compute_end + shared_timing.store_time * 1000;
    float shared_kernel_end = shared_kernel_start + shared_timing.kernel_time * 1000;
    float shared_copy_out_end = shared_kernel_end + shared_timing.copy_out_time * 1000;
    
    std::cout << "PHASE,Shared,H2D_Copy," << shared_start << "," << shared_copy_in_end << "\n";
    std::cout << "PHASE,Shared,Global_to_Shared," << shared_kernel_start << "," << shared_global_to_shared_end << "\n";
    std::cout << "PHASE,Shared,Shared_to_Frag," << shared_global_to_shared_end << "," << shared_shared_to_frag_end << "\n";
    std::cout << "PHASE,Shared,Compute," << shared_shared_to_frag_end << "," << shared_compute_end << "\n";
    std::cout << "PHASE,Shared,Store," << shared_compute_end << "," << shared_store_end << "\n";
    std::cout << "PHASE,Shared,D2H_Copy," << shared_kernel_end << "," << shared_copy_out_end << "\n";
    
    std::cout << "=== TIMELINE_DATA_END ===\n";
}

int main() {
    // CUDA対応チェック
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0) {
        printf("No CUDA-capable devices found\n");
        return 1;
    }
    
    // デバイス情報の取得
    hipDeviceProp_t deviceProp;
    CUDA_CHECK(hipGetDeviceProperties(&deviceProp, 0));
    printf("Using GPU: %s (Compute Capability %d.%d)\n", 
           deviceProp.name, deviceProp.major, deviceProp.minor);
    
    // GPUクロック周波数の取得
    printf("GPU Clock Rate: %.2f GHz\n", deviceProp.clockRate / 1000000.0f);
    
    // TensorCoreサポートの確認
    bool hasTensorCore = (deviceProp.major >= 7);
    if (!hasTensorCore) {
        printf("Warning: This GPU does not support Tensor Cores (requires SM 7.0+)\n");
    } else {
        printf("Tensor Core support detected!\n");
    }
    
    // メモリ確保
    half *A = (half*)malloc(M * K * sizeof(half));
    half *B = (half*)malloc(K * N * sizeof(half));
    float *C_tc = (float*)malloc(M * N * sizeof(float));
    float *C_tc_shared = (float*)malloc(M * N * sizeof(float));
    float *C_ref = (float*)malloc(M * N * sizeof(float));
    
    // 行列の初期化
    printf("Initializing matrices...\n");
    init_matrices(A, B, C_ref);
    
    // CPU上での計算（リファレンス）
    printf("Computing reference solution on CPU...\n");
    double cpu_time = cpu_matmul(A, B, C_ref);
    
    // GPUのウォームアップ
    printf("Warming up GPU...\n");
    TimingInfo warmup1 = profile_tensor_core_global(A, B, C_tc);
    TimingInfo warmup2 = profile_tensor_core_shared(A, B, C_tc_shared);
    
    // ベンチマーク実行回数
    const int NUM_RUNS = 5;
    TimingInfo global_timing = {0};
    TimingInfo shared_timing = {0};
    
    // CUDAカーネルを複数回実行して平均を取る
    printf("Running benchmarks (%d iterations)...\n", NUM_RUNS);
    for (int i = 0; i < NUM_RUNS; i++) {
        printf("  Run %d/%d\n", i+1, NUM_RUNS);
        
        // Tensor Core (グローバルメモリ)
        TimingInfo timing_global = profile_tensor_core_global(A, B, C_tc);
        global_timing.total_time += timing_global.total_time;
        global_timing.copy_in_time += timing_global.copy_in_time;
        global_timing.kernel_time += timing_global.kernel_time;
        global_timing.global_to_frag_time += timing_global.global_to_frag_time;
        global_timing.compute_time += timing_global.compute_time;
        global_timing.store_time += timing_global.store_time;
        global_timing.copy_out_time += timing_global.copy_out_time;
        
        sleep(1);
        
        // Tensor Core (共有メモリ)
        TimingInfo timing_shared = profile_tensor_core_shared(A, B, C_tc_shared);
        shared_timing.total_time += timing_shared.total_time;
        shared_timing.copy_in_time += timing_shared.copy_in_time;
        shared_timing.kernel_time += timing_shared.kernel_time;
        shared_timing.global_to_shared_time += timing_shared.global_to_shared_time;
        shared_timing.shared_to_frag_time += timing_shared.shared_to_frag_time;
        shared_timing.compute_time += timing_shared.compute_time;
        shared_timing.store_time += timing_shared.store_time;
        shared_timing.copy_out_time += timing_shared.copy_out_time;
        shared_timing.sync_overhead += timing_shared.sync_overhead;
        
        sleep(1);
    }
    
    // 平均計算
    global_timing.total_time /= NUM_RUNS;
    global_timing.copy_in_time /= NUM_RUNS;
    global_timing.kernel_time /= NUM_RUNS;
    global_timing.global_to_frag_time /= NUM_RUNS;
    global_timing.compute_time /= NUM_RUNS;
    global_timing.store_time /= NUM_RUNS;
    global_timing.copy_out_time /= NUM_RUNS;
    
    shared_timing.total_time /= NUM_RUNS;
    shared_timing.copy_in_time /= NUM_RUNS;
    shared_timing.kernel_time /= NUM_RUNS;
    shared_timing.global_to_shared_time /= NUM_RUNS;
    shared_timing.shared_to_frag_time /= NUM_RUNS;
    shared_timing.compute_time /= NUM_RUNS;
    shared_timing.store_time /= NUM_RUNS;
    shared_timing.copy_out_time /= NUM_RUNS;
    shared_timing.sync_overhead /= NUM_RUNS;
    
    // 結果の検証
    printf("Validating results...\n");
    bool tc_passed = validate_results(C_tc, C_ref);
    bool tc_shared_passed = validate_results(C_tc_shared, C_ref);
    
    // 結果表示
    printf("\n===== Detailed Performance Results (clock64()を使用) =====\n");
    printf("Matrix Size: %d x %d x %d\n", M, N, K);
    printf("CPU Time: %.6f seconds\n\n", cpu_time);
    
    printf("--- Tensor Core (Global Memory) %s ---\n", tc_passed ? "(PASSED)" : "(FAILED)");
    printf("Total Time: %.6f seconds\n", global_timing.total_time);
    printf("  Copy H->D:     %.6f seconds (%.1f%%)\n", global_timing.copy_in_time, 
           (global_timing.copy_in_time / global_timing.total_time) * 100);
    printf("  Kernel Total:  %.6f seconds (%.1f%%)\n", global_timing.kernel_time, 
           (global_timing.kernel_time / global_timing.total_time) * 100);
    printf("    Global->Frag:  %.9f seconds\n", global_timing.global_to_frag_time);
    printf("    Compute:       %.9f seconds\n", global_timing.compute_time);
    printf("    Store:         %.9f seconds\n", global_timing.store_time);
    printf("  Copy D->H:     %.6f seconds (%.1f%%)\n", global_timing.copy_out_time, 
           (global_timing.copy_out_time / global_timing.total_time) * 100);
    
    printf("\n--- Tensor Core (Shared Memory) %s ---\n", tc_shared_passed ? "(PASSED)" : "(FAILED)");
    printf("Total Time: %.6f seconds\n", shared_timing.total_time);
    printf("  Copy H->D:     %.6f seconds (%.1f%%)\n", shared_timing.copy_in_time, 
           (shared_timing.copy_in_time / shared_timing.total_time) * 100);
    printf("  Kernel Total:  %.6f seconds (%.1f%%)\n", shared_timing.kernel_time, 
           (shared_timing.kernel_time / shared_timing.total_time) * 100);
    printf("    Global->Shared:  %.9f seconds\n", shared_timing.global_to_shared_time);
    printf("    Shared->Frag:    %.9f seconds\n", shared_timing.shared_to_frag_time);
    printf("    Compute:         %.9f seconds\n", shared_timing.compute_time);
    printf("    Store:           %.9f seconds\n", shared_timing.store_time);
    printf("    Sync Overhead:   %.9f seconds\n", shared_timing.sync_overhead);
    printf("  Copy D->H:     %.6f seconds (%.1f%%)\n", shared_timing.copy_out_time, 
           (shared_timing.copy_out_time / shared_timing.total_time) * 100);
    
    // フェーズごとの比較
    printf("\n===== Phase Comparison (Global vs Shared) =====\n");
    printf("Total Kernel Time: %.2fx slower with shared memory\n", 
           shared_timing.kernel_time / global_timing.kernel_time);
    
    // データロード比較（Global->Fragと、Global->Shared+Shared->Fragの比較）
    float global_load_time = global_timing.global_to_frag_time;
    float shared_load_time = shared_timing.global_to_shared_time + shared_timing.shared_to_frag_time;
    printf("Data Loading:      %.2fx slower with shared memory\n", 
           shared_load_time / global_load_time);
    
    printf("Computation:       %.2fx different with shared memory\n", 
           shared_timing.compute_time / global_timing.compute_time);
    printf("Result Storage:    %.2fx different with shared memory\n", 
           shared_timing.store_time / global_timing.store_time);
    
    // 詳細な性能分析
    printf("\n===== Detailed Analysis =====\n");
    
    // 1. 共有メモリオーバーヘッド分析
    float shared_overhead = shared_load_time - global_load_time;
    printf("Shared Memory Overhead: %.9f seconds\n", shared_overhead);
    printf("  - Global->Shared:     %.9f seconds\n", shared_timing.global_to_shared_time);
    printf("  - Shared->Frag:       %.9f seconds\n", shared_timing.shared_to_frag_time);
    printf("  - Direct Global->Frag: %.9f seconds\n", global_timing.global_to_frag_time);
    
    // 2. 同期オーバーヘッド分析
    printf("Synchronization Overhead: %.9f seconds (%.1f%% of shared kernel time)\n", 
           shared_timing.sync_overhead, 
           (shared_timing.sync_overhead / shared_timing.kernel_time) * 100);
    
    // 3. 理論的なTeraFLOPS計算（FP16入力、FP32蓄積のTensor Core演算）
    float operations = 2.0f * M * N * K; // 行列乗算の演算数（乗算+加算）
    float teraflops_global = operations / (global_timing.compute_time * 1e12);
    float teraflops_shared = operations / (shared_timing.compute_time * 1e12);
    printf("Compute Performance:\n");
    printf("  - Global Memory: %.2f TFLOPS\n", teraflops_global);
    printf("  - Shared Memory: %.2f TFLOPS\n", teraflops_shared);

    // タイムラインデータの出力を追加
    output_timeline_data(global_timing, shared_timing);
    
    // リソース解放
    free(A);
    free(B);
    free(C_tc);
    free(C_tc_shared);
    free(C_ref);
    
    return 0;
}